#include "hip/hip_runtime.h"

// Assume we have 256 threads per Thread Block
__host__
int nblocks = (n + 255) / 256;
daxpy<<nblocks, 256>>>(n, 2.0, x, y);
__global__
void daxpy(int n, double a, double *x, double *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}
